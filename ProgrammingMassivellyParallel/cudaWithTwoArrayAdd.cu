#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include "hip/hip_runtime.h"

const double EPSILON = 1.0e-15;
const double a = 1.23;
const double b = 2.34;
const double c = 3.57;

__global__ void vecAddKernel(double* A, double* B, double* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

void check(const double* z, const int N) {
    bool has_error = false;
    for (int i = 0; i < N; ++i) {
        if (fabs(z[i] - c) > EPSILON) {
            has_error = true;
        }
    }
    printf("%s\n", has_error ? "Has errors" : "No errors");
}

void vecAdd(double* A, double* B, double* C, int n) {
    double* A_d, * B_d, * C_d;
    int size = n * sizeof(double);

    hipMalloc((void**)&A_d, size);
    hipMalloc((void**)&B_d, size);
    hipMalloc((void**)&C_d, size);

    hipMemcpy(A_d, A, size, hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, size, hipMemcpyHostToDevice);

    vecAddKernel << <ceil(n / 1024.0), 1024 >> > (A_d, B_d, C_d, n);

    hipMemcpy(C, C_d, size, hipMemcpyDeviceToHost);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
}

int main()
{
    const int N = 10000000;
    double* h_x = (double*)malloc(N * sizeof(double));
    double* h_y = (double*)malloc(N * sizeof(double));
    double* h_z = (double*)malloc(N * sizeof(double));

    for (int i = 0; i < N; ++i) {
        h_x[i] = a;
        h_y[i] = b;
    }

    vecAdd(h_x, h_y, h_z, N);

    check(h_z, N);
}
